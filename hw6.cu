#include <chrono>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <stdio.h>

#define SIZE (1 << 26) // 64M floats, ~256MB

void check(hipError_t err, const char *msg) {
  if (err != hipSuccess) {
    printf("ERROR %s: %s\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}

int main() {
  int dev0 = 0, dev1 = 1;
  size_t bytes = SIZE * sizeof(float);

  // 查询P2P支持
  int canAccessPeer01 = 0, canAccessPeer10 = 0;
  check(hipDeviceCanAccessPeer(&canAccessPeer01, dev0, dev1),
        "hipDeviceCanAccessPeer 0->1");
  check(hipDeviceCanAccessPeer(&canAccessPeer10, dev1, dev0),
        "hipDeviceCanAccessPeer 1->0");
  printf("P2P capability: GPU0->GPU1 %d, GPU1->GPU0 %d\n", canAccessPeer01,
         canAccessPeer10);

  // 1. 准备GPU0内存，初始化数据
  check(hipSetDevice(dev0), "SetDevice 0");
  float *d_data0 = nullptr;
  check(hipMalloc(&d_data0, bytes), "Malloc GPU0");
  float *ori_data = nullptr;
  ori_data = (float *)malloc(SIZE * sizeof(float));
  for (int i = 0; i < SIZE; i++) {
    ori_data[i] = 1.0f;
  }
  check(hipMemcpy(d_data0, ori_data, SIZE * sizeof(float),
                   hipMemcpyHostToDevice),
        "from host copy ori_data to dev0");
  free(ori_data);
  printf("gpu0 data prepared\n");
  // 2. 准备GPU1内存
  check(hipSetDevice(dev1), "SetDevice 1");
  float *d_data1 = nullptr;
  check(hipMalloc(&d_data1, bytes), "Malloc GPU1");
  printf("gpu1 memory prepared\n");

  // 计时辅助
  hipEvent_t start, stop;
  float elapsed_ms;

  // 确保在创建事件前设置成同一个 GPU
  check(hipSetDevice(dev0), "SetDevice 0 (for event creation)");
  check(hipEventCreate(&start), "CreateEvent start");
  check(hipEventCreate(&stop), "CreateEvent stop");
  // 计时辅助

  // --- 1. GPU0->CPU->GPU1 拷贝 ---

  // 准备Host缓冲
  float *h_data = nullptr;
  h_data = (float *)malloc(bytes);

  check(hipSetDevice(dev0), "SetDevice 0");
  check(hipEventRecord(start), "EventRecord start");
  // 从GPU0拷贝到Host
  check(hipMemcpy(h_data, d_data0, bytes, hipMemcpyDeviceToHost),
        "Memcpy D0->H");
  // 切换到GPU1设备
  check(hipSetDevice(dev1), "SetDevice 1");
  // 从Host拷贝到GPU1
  check(hipMemcpy(d_data1, h_data, bytes, hipMemcpyHostToDevice),
        "Memcpy H->D1");
  check(hipEventRecord(stop), "EventRecord stop");
  check(hipEventSynchronize(stop), "EventSynchronize stop");
  check(hipEventElapsedTime(&elapsed_ms, start, stop), "EventElapsedTime");

  printf("GPU0->CPU->GPU1 memcpy time: %.3f ms\n", elapsed_ms);

  free(h_data);

  // --- 2. P2P 拷贝 ---
  // 开启P2P访问
  check(hipSetDevice(dev0), "SetDevice 0");
  if (canAccessPeer01) {
    hipDeviceEnablePeerAccess(dev1, 0);
  }
  check(hipSetDevice(dev1), "SetDevice 1");
  if (canAccessPeer10) {
    hipDeviceEnablePeerAccess(dev0, 0);
  }

  check(hipSetDevice(dev0), "SetDevice 0");
  check(hipEventRecord(start), "EventRecord start");
  // 使用cudaMemcpyPeer从GPU0到GPU1
  check(hipMemcpyPeer(d_data1, dev1, d_data0, dev0, bytes),
        "MemcpyPeer D0->D1");
  check(hipEventRecord(stop), "EventRecord stop");
  check(hipEventSynchronize(stop), "EventSynchronize stop");
  check(hipEventElapsedTime(&elapsed_ms, start, stop), "EventElapsedTime");

  printf("P2P memcpyPeer time: %.3f ms\n", elapsed_ms);

  // 关闭P2P访问
  check(hipSetDevice(dev0), "SetDevice 0");
  if (canAccessPeer01) {
    hipDeviceDisablePeerAccess(dev1);
  }
  check(hipSetDevice(dev1), "SetDevice 1");
  if (canAccessPeer10) {
    hipDeviceDisablePeerAccess(dev0);
  }

  return 0;
}
