#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void twice(float *devicePtr, int N) {

  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  if (tid < N) {
    devicePtr[tid] *= 2.0f;
  }
}

int main() {

  // get total global memory of device 0
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  unsigned long global_mem = prop.totalGlobalMem;
  printf("total global mem :%lu\n", global_mem);

  float *hostPtr;
  float *devicePtr;
  float *data;
  // destribute host memory
  int N = 2 * global_mem / sizeof(float);
  hipMallocManaged((void **)&data, N * sizeof(float));
  // hipHostAlloc((void **)&hostPtr, N * sizeof(float), hipHostMallocDefault);
  for (int i = 0; i < 2 * global_mem / sizeof(float); i += 1) {
    hostPtr[i] = i;
  }

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("hipHostAlloc failed: %s\n", hipGetErrorString(err));
  }

  for (int i = 0; i < 10; i += 1) {
    printf("%f ", hostPtr[i]);
  }
  printf("\n");

  // get map hostPtr to devicePtr
  // hipHostGetDevicePointer((void **)&devicePtr, (void **)&hostPtr, 0);

  // call kernel "twice"
  int grid_size = (N + 1024 - 1) / 1024;
  // twice<<<grid_size, 1024>>>(devicePtr);
  twice<<<grid_size, 1024>>>(data);
  hipDeviceSynchronize();

  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    printf("CUDA error: %s\n", hipGetErrorString(error));
  }

  for (int i = 0; i < 10; i += 1) {
    printf("%f ", hostPtr[i]);
  }
  printf("\n");

  return 0;
}
