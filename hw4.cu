#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void twice(float *devicePtr) {
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  devicePtr[tid] *= 2;
}

int main() {

  // get total global memory of device 0
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  unsigned long global_mem = prop.totalGlobalMem;
  printf("total global mem :%lu\n", global_mem);

  float *hostPtr;
  float *devicePtr;

  // destribute host memory
  int N = 2 * global_mem / sizeof(float);
  hipHostAlloc((void **)&hostPtr, N * sizeof(float), hipHostMallocDefault);
  for (int i = 0; i < 2 * global_mem / sizeof(float); i += 1) {
    hostPtr[i] = i;
  }

  for (int i = 0; i < 10; i += 1) {
    printf("%f ", hostPtr[i]);
  }
  printf("\n");

  // get map hostPtr to devicePtr
  hipHostGetDevicePointer((void **)&devicePtr, (void **)&hostPtr, 0);

  // call kernel "twice"
  int grid_size = (N + 1024 - 1) / 1024;
  twice<<<grid_size, 1024>>>(devicePtr);
  hipDeviceSynchronize();

  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    printf("CUDA error: %s\n", hipGetErrorString(error));
  }

  for (int i = 0; i < 10; i += 1) {
    printf("%f ", hostPtr[i]);
  }
  printf("\n");

  return 0;
}
