#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void square(int *ori, int *target, int size) {
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (idx < size) {
    target[idx] = ori[idx] * ori[idx];
  }
}

int main() {
  // ptr on host
  int size = 1e5;
  int *A = (int *)malloc(size * sizeof(int));
  int *res = (int *)malloc(size * sizeof(int));
  // ptr on device
  int *d_A;
  int *d_B;
  hipMalloc((int **)&d_A, size * sizeof(int));
  hipMalloc((int **)&d_B, size * sizeof(int));

  // initialize arr A on host
  for (int i = 0; i < size; i++) {
    A[i] = i % 10 + 1;
  }
  // copy data to device
  hipMemcpy(d_A, A, size * sizeof(int), hipMemcpyHostToDevice);

  // cuda events
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start, 0);
  // start kernel
  int block_size = 1024;
  int grid_size = (size + block_size - 1) / block_size;
  square<<<grid_size, block_size>>>(d_A, d_B, size);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  float time_cost;
  hipEventElapsedTime(&time_cost, start, stop);

  // copy back to host
  hipMemcpy(res, d_B, size * sizeof(int), hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_B);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  // print the first 15 numbers of result
  for (int i = 0; i < 15; i++) {
    printf("%d,", res[i]);
  }
  printf("\n");
  printf("Time cost:%f\n", time_cost);
}
